
// Standard utilities and common systems includes
#include "kernels.cuh"
#include "cpu_functions.c"
#include "hipcub/hipcub.hpp"

//#define DIMENSION 1000


////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////

//uint64_t filter_hits(hit_advanced * hits_in, ulong kmer_size, ulong n_hits_found);

int main(int argc, char ** argv)
{
    uint64_t i;
    unsigned selected_device = 0, write = 0;
    FILE * query = NULL, * ref = NULL, * out = NULL;
    init_args(argc, argv, &query, &selected_device, &ref, &out, &write);

    ////////////////////////////////////////////////////////////////////////////////
    // Get info of devices
    ////////////////////////////////////////////////////////////////////////////////

    int ret_num_devices;
    unsigned compute_units;
    uint64_t local_device_RAM, global_device_RAM;
    int work_group_dimensions[3], work_group_size_local;
    int ret;
    
    // Query how many devices there are
    if(hipSuccess != (ret = hipGetDeviceCount(&ret_num_devices))){ fprintf(stderr, "Failed to query number of devices\n"); exit(-1); }

    hipDeviceProp_t device;

    for(i=0; i<ret_num_devices; i++){
        if( hipSuccess != (ret = hipGetDeviceProperties(&device, i))){ fprintf(stderr, "Failed to get cuda device property: %d\n", ret); exit(-1); }

        fprintf(stdout, "\tDevice [%"PRIu64"]: %s\n", i, device.name);
        global_device_RAM = device.totalGlobalMem;
        fprintf(stdout, "\t\tGlobal mem   : %"PRIu64" (%"PRIu64" MB)\n", (uint64_t) global_device_RAM, (uint64_t) global_device_RAM / (1024*1024));
        local_device_RAM = device.sharedMemPerBlock;
        fprintf(stdout, "\t\tLocal mem    : %"PRIu64" (%"PRIu64" KB)\n", (uint64_t) local_device_RAM, (uint64_t) local_device_RAM / (1024));
        compute_units = device.multiProcessorCount;
        fprintf(stdout, "\t\tCompute units: %"PRIu64"\n", (uint64_t) compute_units);
        work_group_size_local = device.maxThreadsPerBlock;
        fprintf(stdout, "\t\tMax work group size: %d\n", work_group_size_local);
        memcpy(work_group_dimensions, device.maxThreadsDim, 3*sizeof(int));
        fprintf(stdout, "\t\tWork size dimensions: (%d, %d, %d)\n", work_group_dimensions[0], work_group_dimensions[1], work_group_dimensions[2]);
        fprintf(stdout, "\t\tWarp size: %d\n", device.warpSize);
        fprintf(stdout, "\t\tGrid dimensions: (%d, %d, %d)\n", device.maxGridSize[0], device.maxGridSize[1], device.maxGridSize[2]);
    }
    //selected_device = 3; // REMOVE --- ONLY FOR TESTING $$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$$

    if( hipSuccess != (ret = hipSetDevice(selected_device))){ fprintf(stderr, "Failed to get cuda device property: %d\n", ret); exit(-1); }
    fprintf(stdout, "[INFO] Using device %d\n", selected_device);

    if( hipSuccess != (ret = hipGetDeviceProperties(&device, selected_device))){ fprintf(stderr, "Failed to get cuda device property: %d\n", ret); exit(-1); }
    global_device_RAM = device.totalGlobalMem;

    
    
    ////////////////////////////////////////////////////////////////////////////////
    // Make index dictionary
    ////////////////////////////////////////////////////////////////////////////////

    

    // Calculate how much ram we can use for every chunk
    uint64_t effective_global_ram =  (global_device_RAM - 100*1024*1024); //Minus 100 MBs
    uint64_t ram_to_be_used = (effective_global_ram) / (2 * (sizeof(Word) + sizeof(char))); //
    uint64_t words_at_once = ram_to_be_used;


    uint64_t * keys, * values, * keys_buf, * values_buf;
    fprintf(stdout, "[INFO] I will use %"PRIu64" bytes for hash (you can have %"PRIu64" entries) and their buffers\n", words_at_once * 2 * sizeof(Word), words_at_once);

    

    
    // Set working size
    size_t threads_number = 32;
    size_t number_of_blocks;
    //hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte); // NOTICE: MAXWELL ignores this--

    

    // Inspect shared memory configuration
    hipSharedMemConfig shared_mem_conf;
    ret = hipDeviceGetSharedMemConfig(&shared_mem_conf);
    if(ret != hipSuccess){ fprintf(stdout, "[WARNING] Could not get shared memory configuration. Error: %d\n", ret); }
    else { fprintf(stdout, "[INFO] Shared memory configuration is: %s\n", (shared_mem_conf == hipSharedMemBankSizeFourByte) ? ("4 bytes") : ("8 bytes")); }

    // Load DNA sequences
    uint64_t query_len = get_seq_len(query);
    uint64_t ref_len = get_seq_len(ref);

    char * query_seq_host = (char *) malloc(query_len * sizeof(char));
    char * ref_seq_host = (char *) malloc(ref_len * sizeof(char));

    if(query_seq_host == NULL || ref_seq_host == NULL) terror("Could not allocate memory for sequences in host");

    fprintf(stdout, "[INFO] Loading query\n");
    load_seq(query, query_seq_host);
    fprintf(stdout, "[INFO] Loading reference\n");
    load_seq(ref, ref_seq_host);

    
     
    
    clock_t begin;

    // Pointer to device memory allocating the query sequence
    char * seq_dev_mem = NULL;


    // Allocate memory in host to download kmers and store hits
    uint64_t * dict_x_keys, * dict_x_values, * dict_y_keys, * dict_y_values;
    dict_x_keys = (uint64_t *) malloc(words_at_once*sizeof(uint64_t));
    dict_x_values = (uint64_t *) malloc(words_at_once*sizeof(uint64_t));
    if(dict_x_keys == NULL || dict_x_values == NULL) { fprintf(stderr, "Allocating for kmer download in query. Error: %d\n", ret); exit(-1); }
    dict_y_keys = (uint64_t *) malloc(words_at_once*sizeof(uint64_t));
    dict_y_values = (uint64_t *) malloc(words_at_once*sizeof(uint64_t));
    if(dict_y_keys == NULL || dict_y_values == NULL) { fprintf(stderr, "Allocating for kmer download in ref. Error: %d\n", ret); exit(-1); }
    Hit * hits = (Hit *) malloc(words_at_once*sizeof(Hit));
    Hit * filtered_hits = (Hit *) malloc(words_at_once*sizeof(Hit));
    if(hits == NULL || filtered_hits == NULL) terror("Could not allocate hits");
    uint64_t * diagonals = (uint64_t *) malloc(words_at_once*sizeof(uint64_t));
    uint64_t * device_diagonals, * device_diagonals_buf;
    uint64_t * device_hits, * device_hits_buf; // These will actually be just indices to redirect the hits sorting
    uint64_t * ascending_numbers = (uint64_t *) malloc(words_at_once*sizeof(uint64_t)); for(i=0; i<words_at_once; i++) ascending_numbers[i] = i;
    uint64_t * indexing_numbers = (uint64_t *) malloc(words_at_once*sizeof(uint64_t));
    if(hits == NULL) { fprintf(stderr, "Allocating for hits download. Error: %d\n", ret); exit(-1); }


    ////////////////////////////////////////////////////////////////////////////////
    // Read the query and reference in blocks
    ////////////////////////////////////////////////////////////////////////////////

    
    fprintf(out, "All by-Identity Ungapped Fragments (Hits based approach)\n");
    fprintf(out, "[Abr.98/Apr.2010/Dec.2011 -- <ortrelles@uma.es>\n");
    fprintf(out, "SeqX filename : undef\n");
    fprintf(out, "SeqY filename : undef\n");
    fprintf(out, "SeqX name : undef\n");
    fprintf(out, "SeqY name : undef\n");
    fprintf(out, "SeqX length : %"PRIu64"\n", query_len);
    fprintf(out, "SeqY length : %"PRIu64"\n", ref_len);
    fprintf(out, "Min.fragment.length : undef\n");
    fprintf(out, "Min.Identity : undef\n");
    fprintf(out, "Tot Hits (seeds) : undef\n");
    fprintf(out, "Tot Hits (seeds) used: undef\n");
    fprintf(out, "Total fragments : undef\n");
    fprintf(out, "========================================================\n");
    fprintf(out, "Total CSB: 0\n");
    fprintf(out, "========================================================\n");
    fprintf(out, "Type,xStart,yStart,xEnd,yEnd,strand(f/r),block,length,score,ident,similarity,%%ident,SeqX,SeqY\n");

    int split = 0;
    uint64_t pos_in_query = 0, pos_in_ref = 0;
    while(pos_in_query < query_len){

        // Allocate memory in device for sequence chunk
        // We have to this here since later on we will have to free all memory to load the hits
        ret = hipMalloc(&seq_dev_mem, words_at_once * sizeof(char));
        if(ret != hipSuccess){ fprintf(stderr, "Could not allocate memory for query sequence in device (Attempted %"PRIu64" bytes). Error: %d\n", words_at_once * sizeof(char), ret); exit(-1); }



        // Allocate words table
        ret = hipMalloc(&keys, words_at_once * sizeof(uint64_t));
        if(ret != hipSuccess){ fprintf(stderr, "Could not allocate memory for table in device (1). Error: %d\n", ret); exit(-1); }
        ret = hipMalloc(&values, words_at_once * sizeof(uint64_t));
        if(ret != hipSuccess){ fprintf(stderr, "Could not allocate memory for table in device (2). Error: %d\n", ret); exit(-1); }
        ret = hipMalloc(&keys_buf, words_at_once * sizeof(uint64_t));
        if(ret != hipSuccess){ fprintf(stderr, "Could not allocate memory for table in device (3). Error: %d\n", ret); exit(-1); }
        ret = hipMalloc(&values_buf, words_at_once * sizeof(uint64_t));
        if(ret != hipSuccess){ fprintf(stderr, "Could not allocate memory for table in device (4). Error: %d\n", ret); exit(-1); }

        fprintf(stdout, "[EXECUTING] Running split %d -> (%d%%)\n", split, (int)((100*pos_in_query)/query_len));
        uint64_t items_read = MIN(query_len - pos_in_query, words_at_once);


        ////////////////////////////////////////////////////////////////////////////////
        // Run kmers for query
        ////////////////////////////////////////////////////////////////////////////////
        
        // Load sequence chunk into ram
        ret = hipMemcpy(seq_dev_mem, &query_seq_host[pos_in_query], items_read, hipMemcpyHostToDevice);
        if(ret != hipSuccess){ fprintf(stderr, "Could not copy query sequence to device. Error: %d\n", ret); exit(-1); }

        // Run kmers
        number_of_blocks = (((items_read - KMER_SIZE + 1)) / (threads_number*4));
        ret = hipMemset(keys, 0xFFFFFFFF, words_at_once * sizeof(uint64_t));
        ret = hipMemset(values, 0xFFFFFFFF, words_at_once * sizeof(uint64_t));
        kernel_register_fast_hash_rotational<<<number_of_blocks, threads_number>>>(keys, values, seq_dev_mem, pos_in_query);
        ret = hipDeviceSynchronize();
        if(ret != hipSuccess){ fprintf(stderr, "Could not compute kmers on query. Error: %d\n", ret); exit(-1); }

        ////////////////////////////////////////////////////////////////////////////////
        // Sort the query kmers
        ////////////////////////////////////////////////////////////////////////////////

        hipcub::DoubleBuffer<uint64_t> d_keys(keys, keys_buf);
        hipcub::DoubleBuffer<uint64_t> d_values(values, values_buf);
        void * d_temp_storage = NULL;
        size_t temp_storage_bytes = 0;
        hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, items_read);
        ret = hipDeviceSynchronize();
        if(ret != hipSuccess){ fprintf(stderr, "Bad pre-sorting (1). Error: %d -> %s\n", ret, hipGetErrorString(hipGetLastError())); exit(-1); }

        // Allocate temporary storage
        ret = hipMalloc(&d_temp_storage, temp_storage_bytes);
        if(ret != hipSuccess){ fprintf(stderr, "Bad allocating of temp storage for words sorting (1). Error: %d -> %s\n", ret, hipGetErrorString(hipGetLastError())); exit(-1); }

        // Remove this for  debug
        //read_kmers(query_len, query_seq_host, dict_x_keys, dict_x_values);
        //ret = hipMemcpy(keys, dict_x_keys, items_read*sizeof(uint64_t), hipMemcpyHostToDevice);
        //ret = hipMemcpy(values, dict_x_values, items_read*sizeof(uint64_t), hipMemcpyHostToDevice);
        
        hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys, d_values, items_read);
        ret = hipDeviceSynchronize();
        if(ret != hipSuccess){ fprintf(stderr, "CUB sorting failed on query. Error: %d -> %s\n", ret, hipGetErrorString(hipGetLastError())); exit(-1); }
        

        // Download kmers        
        ret = hipMemcpy(dict_x_keys, keys, items_read*sizeof(uint64_t), hipMemcpyDeviceToHost);
        if(ret != hipSuccess){ fprintf(stderr, "Downloading device kmers (1). Error: %d\n", ret); exit(-1); }
        ret = hipMemcpy(dict_x_values, values, items_read*sizeof(uint64_t), hipMemcpyDeviceToHost);
        if(ret != hipSuccess){ fprintf(stderr, "Downloading device kmers (2). Error: %d\n", ret); exit(-1); }

        // Print hits for debug
        //for(i=0; i<items_read; i++){
        //    fprintf(out, "%"PRIu64"\n", dict_x_values[i]);
        //}

        ret = hipFree(d_temp_storage);
        if(ret != hipSuccess){ fprintf(stderr, "Bad free of temp storage (1): %d -> %s\n", ret, hipGetErrorString(hipGetLastError())); exit(-1); }

        pos_in_query += words_at_once;

        hipFree(seq_dev_mem);
        hipFree(keys);
        hipFree(values);
        hipFree(keys_buf);
        hipFree(values_buf);        

        ////////////////////////////////////////////////////////////////////////////////
        // Run the reference blocks
        ////////////////////////////////////////////////////////////////////////////////

        while(pos_in_ref < ref_len){

            items_read = MIN(ref_len - pos_in_ref, words_at_once);

            ret = hipMalloc(&seq_dev_mem, words_at_once * sizeof(char));

            // Allocate words table
            ret = hipMalloc(&keys, words_at_once * sizeof(uint64_t));
            if(ret != hipSuccess){ fprintf(stderr, "Could not allocate memory for table in device (1). Error: %d\n", ret); exit(-1); }
            ret = hipMalloc(&values, words_at_once * sizeof(uint64_t));
            if(ret != hipSuccess){ fprintf(stderr, "Could not allocate memory for table in device (2). Error: %d\n", ret); exit(-1); }
            ret = hipMalloc(&keys_buf, words_at_once * sizeof(uint64_t));
            if(ret != hipSuccess){ fprintf(stderr, "Could not allocate memory for table in device (3). Error: %d\n", ret); exit(-1); }
            ret = hipMalloc(&values_buf, words_at_once * sizeof(uint64_t));
            if(ret != hipSuccess){ fprintf(stderr, "Could not allocate memory for table in device (4). Error: %d\n", ret); exit(-1); }

            // Load sequence chunk into ram
            ret = hipMemcpy(seq_dev_mem, &ref_seq_host[pos_in_ref], items_read, hipMemcpyHostToDevice);
            if(ret != hipSuccess){ fprintf(stderr, "Could not copy ref sequence to device. Error: %d\n", ret); exit(-1); }

            // Run kmers
            number_of_blocks = (((items_read - KMER_SIZE + 1)) / (threads_number*4)); 
            ret = hipMemset(keys, 0xFFFFFFFF, words_at_once * sizeof(uint64_t));
            ret = hipMemset(values, 0xFFFFFFFF, words_at_once * sizeof(uint64_t));
            kernel_register_fast_hash_rotational<<<number_of_blocks, threads_number>>>(keys, values, seq_dev_mem, pos_in_ref);
            ret = hipDeviceSynchronize();
            if(ret != hipSuccess){ fprintf(stderr, "Could not compute kmers on ref. Error: %d\n", ret); exit(-1); }

            // remove all this             
            
            //read_kmers(ref_len, ref_seq_host, dict_y_keys, dict_y_values);
            //ret = hipMemcpy(keys, dict_y_keys, items_read*sizeof(uint64_t), hipMemcpyHostToDevice);
            //ret = hipMemcpy(values, dict_y_values, items_read*sizeof(uint64_t), hipMemcpyHostToDevice);


            ////////////////////////////////////////////////////////////////////////////////
            // Sort reference kmers
            ////////////////////////////////////////////////////////////////////////////////

            ret = hipMemset(keys_buf, 0xFFFFFFFF, words_at_once * sizeof(uint64_t));
            ret = hipMemset(values_buf, 0xFFFFFFFF, words_at_once * sizeof(uint64_t));
            hipcub::DoubleBuffer<uint64_t> d_keys_ref(keys, keys_buf);
            hipcub::DoubleBuffer<uint64_t> d_values_ref(values, values_buf);
            d_temp_storage = NULL;
            temp_storage_bytes = 0;
            hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_ref, d_values_ref, items_read);
            ret = hipDeviceSynchronize();
            if(ret != hipSuccess){ fprintf(stderr, "Bad pre-sorting (2). Error: %d -> %s\n", ret, hipGetErrorString(hipGetLastError())); exit(-1); }

            // Allocate temporary storage
            ret = hipMalloc(&d_temp_storage, temp_storage_bytes);
            if(ret != hipSuccess){ fprintf(stderr, "Bad allocating of temp storage for words sorting (2). Error: %d -> %s\n", ret, hipGetErrorString(hipGetLastError())); exit(-1); }
            
            hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_keys_ref, d_values_ref, items_read);
            ret = hipDeviceSynchronize();
            if(ret != hipSuccess){ fprintf(stderr, "CUB sorting failed on ref. Error: %d -> %s\n", ret, hipGetErrorString(hipGetLastError())); exit(-1); }
            


            // Download kmers
            ret = hipMemcpy(dict_y_keys, keys, items_read*sizeof(uint64_t), hipMemcpyDeviceToHost);
            if(ret != hipSuccess){ fprintf(stderr, "Downloading device kmers (3). Error: %d\n", ret); exit(-1); }
            ret = hipMemcpy(dict_y_values, values, items_read*sizeof(uint64_t), hipMemcpyDeviceToHost);
            if(ret != hipSuccess){ fprintf(stderr, "Downloading device kmers (4). Error: %d\n", ret); exit(-1); }

            ret = hipFree(d_temp_storage);
            if(ret != hipSuccess){ fprintf(stderr, "Bad free of temp storage (2): %d -> %s\n", ret, hipGetErrorString(hipGetLastError())); exit(-1); }

            pos_in_ref += words_at_once;


            ////////////////////////////////////////////////////////////////////////////////
            // Generate hits for the current split
            ////////////////////////////////////////////////////////////////////////////////

            
            //read_kmers(query_len, query_seq_host, dict_x_keys, dict_x_values);
            //Qsort(dict_x_keys, dict_x_values, 0, (int64_t) query_len);
            //for(i=0; i<words_at_once; i++) printf("%" PRIu64" %"PRIu64"\n", dict_x_keys[i], dict_x_values[i]);
            //read_kmers(ref_len, ref_seq_host, dict_y_keys, dict_y_values);
            //Qsort(dict_y_keys, dict_y_values, 0, (int64_t) ref_len);
            //for(i=0; i<words_at_once; i++) printf("%" PRIu64" %"PRIu64"\n", dict_x_keys[i], dict_x_values[i]);

            hipFree(seq_dev_mem);
            hipFree(keys);
            hipFree(values);
            hipFree(keys_buf);
            hipFree(values_buf);


            uint64_t n_hits_found = generate_hits(words_at_once, diagonals, hits, dict_x_keys, dict_y_keys, dict_x_values, dict_y_values, query_len, ref_len);
            
            fprintf(stdout, "[INFO] Generated %"PRIu64" hits on split %d -> (%d%%)\n", n_hits_found, split, (int)((100*MIN(pos_in_ref, ref_len))/ref_len));

            // Print hits for debug
            //for(i=0; i<items_read; i++){
                //fprintf(out, "%"PRIu64"\n", dict_x_values[i]);
            //}
            //for(i=0; i<n_hits_found; i++){
                //printf("%"PRIu64"\n", diagonals[i]);
                //fprintf(out, "Frag,d:%"PRIu64",%"PRIu64",%"PRIu64",%"PRIu64",%"PRIu64",f,0,32,32,32,1.0,1.0,0,0\n", diagonals[i], hits[i].p1, hits[i].p2, hits[i].p1+32, hits[i].p2+32);
            //}

            ////////////////////////////////////////////////////////////////////////////////
            // Sort hits for the current split
            ////////////////////////////////////////////////////////////////////////////////

            ret = hipMalloc(&device_diagonals, words_at_once * sizeof(uint64_t));
            if(ret != hipSuccess){ fprintf(stderr, "Could not allocate memory for hits in device (1). Error: %d\n", ret); exit(-1); }
            ret = hipMalloc(&device_diagonals_buf, words_at_once * sizeof(uint64_t));
            if(ret != hipSuccess){ fprintf(stderr, "Could not allocate memory for hits in device (2). Error: %d\n", ret); exit(-1); }
            ret = hipMalloc(&device_hits, words_at_once * sizeof(uint64_t));
            if(ret != hipSuccess){ fprintf(stderr, "Could not allocate memory for hits in device (3). Error: %d\n", ret); exit(-1); }
            ret = hipMalloc(&device_hits_buf, words_at_once * sizeof(uint64_t));
            if(ret != hipSuccess){ fprintf(stderr, "Could not allocate memory for hits in device (4). Error: %d\n", ret); exit(-1); }

            // We will actually sort the diagonals with associated values 0,1,2... to n and use these to index the hits array
            ret = hipMemcpy(device_hits, ascending_numbers, n_hits_found*sizeof(uint64_t), hipMemcpyHostToDevice);
            if(ret != hipSuccess){ fprintf(stderr, "Uploading device hits. Error: %d\n", ret); exit(-1); }

            ret = hipMemcpy(device_diagonals, diagonals, n_hits_found*sizeof(uint64_t), hipMemcpyHostToDevice);
            if(ret != hipSuccess){ fprintf(stderr, "Uploading device diagonals. Error: %d\n", ret); exit(-1); }

            hipcub::DoubleBuffer<uint64_t> d_diagonals(device_diagonals, device_diagonals_buf);
            hipcub::DoubleBuffer<uint64_t> d_hits(device_hits, device_hits_buf);
            d_temp_storage = NULL;
            temp_storage_bytes = 0;
            hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_diagonals, d_hits, n_hits_found);
            ret = hipDeviceSynchronize();
            if(ret != hipSuccess){ fprintf(stderr, "Bad pre-sorting (3). Error: %d -> %s\n", ret, hipGetErrorString(hipGetLastError())); exit(-1); }

            // Allocate temporary storage
            ret = hipMalloc(&d_temp_storage, temp_storage_bytes);
            if(ret != hipSuccess){ fprintf(stderr, "Bad allocating of temp storage for hits sorting (1). Error: %d -> %s\n", ret, hipGetErrorString(hipGetLastError())); exit(-1); }
            
            hipcub::DeviceRadixSort::SortPairs(d_temp_storage, temp_storage_bytes, d_diagonals, d_hits, n_hits_found);
            ret = hipDeviceSynchronize();
            if(ret != hipSuccess){ fprintf(stderr, "CUB sorting failed on hits. Error: %d -> %s\n", ret, hipGetErrorString(hipGetLastError())); exit(-1); }
            
            // Download hits (actually just number indices)
            ret = hipMemcpy(indexing_numbers, device_hits, n_hits_found*sizeof(uint64_t), hipMemcpyDeviceToHost);
            if(ret != hipSuccess){ fprintf(stderr, "Downloading device hits. Error: %d\n", ret); exit(-1); }

            ret = hipMemcpy(diagonals, device_diagonals, n_hits_found*sizeof(uint64_t), hipMemcpyDeviceToHost);
            if(ret != hipSuccess){ fprintf(stderr, "Downloading device diagonals. Error: %d\n", ret); exit(-1); }


            uint64_t n_hits_kept = filter_hits(diagonals, indexing_numbers, hits, filtered_hits, n_hits_found);

            fprintf(stdout, "[INFO] Remaining hits %"PRIu64"\n", n_hits_kept);

            for(i=0; i<n_hits_kept; i++){
                //printf("%"PRIu64"\n", diagonals[i]);
                fprintf(out, "Frag,%"PRIu64",%"PRIu64",%"PRIu64",%"PRIu64",f,0,32,32,32,1.0,1.0,0,0\n", filtered_hits[i].p1, filtered_hits[i].p2, filtered_hits[i].p1+32, filtered_hits[i].p2+32);
            }
            
            ret = hipFree(d_temp_storage);
            ret = hipFree(device_hits);
            ret = hipFree(device_diagonals);
            ret = hipFree(device_diagonals_buf);
            ret = hipFree(device_hits_buf);

        }

        // Restart the reference for every block in query
        pos_in_ref = 0;




        ++split;
    }

    fclose(out);

    if(write == 1)
    {
        print_kmers_to_file(dict_x_keys, dict_x_values, query_len, out);
        //print_kmers_to_file_paused(table_mem, query_len_bytes);
    }

    /*
    uint64_t * debug = (uint64_t *) malloc(table_size*sizeof(uint64_t));
    ret = hipMemcpy(debug, table_mem, table_size*sizeof(uint64_t), hipMemcpyDeviceToHost); 
    if(ret != hipSuccess){ fprintf(stderr, "DEBUG. Error: %d\n", ret); exit(-1); }
    
    for(i=0;i<12;i++){
        fprintf(stdout, "#%"PRIu64": %"PRIu64"\n", i, debug[i]);
    } 
    */
    
    
    
    
    

    fprintf(stdout, "[INFO] Completed\n");

    fclose(query);
    fclose(ref);
    free(query_seq_host);
    free(ref_seq_host);
    free(diagonals);
    free(ascending_numbers);
    

    return 0;
}




