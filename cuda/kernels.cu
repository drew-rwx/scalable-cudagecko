#include "hip/hip_runtime.h"
#include "kernels.cuh"

__constant__ uint64_t pow4[33]={1L, 4L, 16L, 64L, 256L, 1024L, 4096L, 16384L, 65536L,
    262144L, 1048576L, 4194304L, 16777216L, 67108864L, 268435456L, 1073741824L, 4294967296L,
    17179869184L, 68719476736L, 274877906944L, 1099511627776L, 4398046511104L, 17592186044416L,
    70368744177664L, 281474976710656L, 1125899906842624L, 4503599627370496L, 18014398509481984L,
	72057594037927936L, 288230376151711744L, 1152921504606846976L, 4611686018427387904L};
	

__constant__ uint64_t pow4_G[33]={2*1L, 2*4L, 2*16L, 2*64L, 2*256L, 2*1024L, 2*4096L, 2*16384L, 2*65536L,
    (uint64_t)2*262144L, (uint64_t)2*1048576L,(uint64_t)2*4194304L, (uint64_t)2*16777216L, (uint64_t)2*67108864L, (uint64_t)2*268435456L, (uint64_t)2*1073741824L, (uint64_t)2*4294967296L,
    (uint64_t)2*17179869184L, (uint64_t)2*68719476736L, (uint64_t)2*274877906944L, (uint64_t)2*1099511627776L, (uint64_t)2*4398046511104L, (uint64_t)2*17592186044416L,
    (uint64_t)2*70368744177664L, (uint64_t)2*281474976710656L, (uint64_t)2*1125899906842624L, (uint64_t)2*4503599627370496L, (uint64_t)2*18014398509481984L,
    (uint64_t)2*72057594037927936L, (uint64_t) 2*288230376151711744L, (uint64_t) 2*1152921504606846976L, (uint64_t) 2*4611686018427387904L};

__constant__ uint64_t pow4_T[33]={3*1L, 3*4L, 3*16L, 3*64L, 3*256L, 3*1024L, 3*4096L, 3*16384L, 3*65536L,
    (uint64_t)3*262144L, (uint64_t) 3*1048576L, (uint64_t)3*4194304L, (uint64_t)3*16777216L, (uint64_t)3*67108864L, (uint64_t)3*268435456L, (uint64_t)3*1073741824L, (uint64_t)3*4294967296L,
    (uint64_t)3*17179869184L, (uint64_t)3*68719476736L, (uint64_t)3*274877906944L, (uint64_t)3*1099511627776L, (uint64_t)3*4398046511104L, (uint64_t)3*17592186044416L,
    (uint64_t)3*70368744177664L, (uint64_t)3*281474976710656L, (uint64_t)3*1125899906842624L, (uint64_t)3*4503599627370496L, (uint64_t)3*18014398509481984L,
    (uint64_t)3*72057594037927936L, (uint64_t) 3*288230376151711744L, (uint64_t) 3*1152921504606846976L, (uint64_t) 3*4611686018427387904L};

#define MIN_P_IDENT 80

__global__ void kernel_frags_forward_register(uint32_t * h_p1, uint32_t * h_p2, uint32_t * left_offset, uint32_t * right_offset, const char * seq_x, const char * seq_y, uint32_t query_len, uint32_t ref_len, uint32_t x_seq_off, uint32_t y_seq_off, uint32_t x_lim, uint32_t y_lim){


	// LEFT ALIGNMENT
	int32_t hp1 = (int32_t) h_p1[blockIdx.x];
	int32_t warp_pos_x_left = hp1;
	int32_t warp_pos_y_left = (int32_t) h_p2[blockIdx.x];
	int32_t thre_pos_x, thre_pos_y;
	uint32_t best_offset_left = (uint32_t) hp1;
	int32_t score = 32, best_score = 32, total_idents = 16; // Half of the hit for each side
	int p_ident = 100;
	int cell_score;

    //int counter = 0;
    //if(threadIdx.x==0)printf("BLOCK %d %d new frag, new life - start %u\n", blockIdx.x, counter++, hp1);
	
	while(score > 0 && (warp_pos_x_left - 32) >= (int32_t) x_seq_off && (warp_pos_y_left - 32) >= (int32_t) y_seq_off)
	{
		
		warp_pos_x_left -= 32;
		warp_pos_y_left -= 32;
		thre_pos_x = warp_pos_x_left + threadIdx.x;
		thre_pos_y = warp_pos_y_left + threadIdx.x;

		

		char v_x = seq_x[thre_pos_x - (int32_t) x_seq_off];
		char v_y = seq_y[thre_pos_y - (int32_t) y_seq_off];

		cell_score = (v_x == v_y);

		for (int offset = 16; offset > 0; offset = offset >> 1)
			cell_score += __shfl_down_sync(0xFFFFFFFF, cell_score, offset);

		
		int idents = __shfl_sync(0xFFFFFFFF, cell_score, 0);
		score = score + (int32_t) idents;
        total_idents += (int32_t) idents;
		score = score - (int32_t) (32 - idents);
		p_ident = ((100 * total_idents) / (int) (hp1 - warp_pos_x_left + 16));
        //if(threadIdx.x == 0) printf("BLOCK %d %d yea current pident to left %d positions: [start: %u and moved to %u]\n", blockIdx.x, counter++, p_ident, hp1, warp_pos_x_left);
		if(score > best_score && p_ident >= MIN_P_IDENT){ best_score = score; best_offset_left = warp_pos_x_left; }
		
		
	}
	
    //if(threadIdx.x==0) printf("BLOCK %d %d end left al\n", blockIdx.x, counter++);	
	
	

	// RIGHT ALIGNMENT
	int32_t warp_pos_x_right = hp1 + 32;
	int32_t warp_pos_y_right = (int32_t) h_p2[blockIdx.x] + 32;
	uint32_t best_offset_right = (uint32_t) warp_pos_x_right;
	score = 32;
    total_idents = 16;
	best_score = 32;
	p_ident = 100;

	
	while(score > 0 && (warp_pos_x_right + 32) < (int32_t) x_lim && (warp_pos_y_right + 32) < (int32_t) y_lim)
	{
		thre_pos_x = warp_pos_x_right + threadIdx.x;
		thre_pos_y = warp_pos_y_right + threadIdx.x;
		char v_x = seq_x[thre_pos_x - (int32_t) x_seq_off];
		char v_y = seq_y[thre_pos_y - (int32_t) y_seq_off];

		//cell_score = (v_x == v_y && v_x != '\0' && v_y != '\0');
		cell_score = (v_x == v_y);

		for (int offset = 16; offset > 0; offset = offset >> 1)
			cell_score += __shfl_down_sync(0xFFFFFFFF, cell_score, offset);
		
		int idents = __shfl_sync(0xFFFFFFFF, cell_score, 0);
		score = score + (int32_t) idents;
        total_idents += (int32_t) idents;
		score = score - (int32_t) (32 - idents);
		p_ident = (int) ((100 * total_idents) / (int32_t) (warp_pos_x_right - (hp1) + 16));

		warp_pos_x_right += 32;
		warp_pos_y_right += 32; 

        //if(threadIdx.x == 0) printf("BLOCK %d %d yea current pident to right %d positions: [start: %u and moved to %u]\n", blockIdx.x, counter++, p_ident, hp1 + 32, warp_pos_x_right);

		if(score > best_score && p_ident >= MIN_P_IDENT){ best_score = score; best_offset_right = warp_pos_x_right; }

	}
	
	
    //if(threadIdx.x==0) printf("BLOCK %d %d end right al\n", blockIdx.x, counter++);	

	// Save at the end
	if(threadIdx.x == 0){
		left_offset[blockIdx.x] = hp1 - (uint32_t) best_offset_left;
        //printf("BLOCK %d %d left offset:  %u\n", blockIdx.x, counter++, left_offset[blockIdx.x]);
		right_offset[blockIdx.x] = (uint32_t) (best_offset_right) - hp1;
        //printf("BLOCK %d %d right offset: %u\n", blockIdx.x, counter++, right_offset[blockIdx.x]);
	}

}


__global__ void kernel_frags_reverse_register(uint32_t * h_p1, uint32_t * h_p2, uint32_t * left_offset, uint32_t * right_offset, const char * seq_x, const char * seq_y, uint32_t query_len, uint32_t ref_len, uint32_t x_seq_off, uint32_t y_seq_off, uint32_t x_lim, uint32_t y_lim){

	// LEFT ALIGNMENT
	int32_t hp1 = (int32_t) h_p1[blockIdx.x];
	int32_t warp_pos_x_left = hp1;
	int32_t warp_pos_y_left = (int32_t) h_p2[blockIdx.x];
	int32_t thre_pos_x, thre_pos_y;
	uint32_t best_offset_left = (uint32_t) hp1;
	int32_t score = 32, best_score = 32, total_idents = 16;
	int p_ident = 100;
	int cell_score;

    //int counter = 0;
    //if(threadIdx.x==0)printf("BLOCK %d %d new frag, new life - startx %u starty %u\n", blockIdx.x, counter++, hp1, h_p2[blockIdx.x]);	

	
	//while(p_ident > MIN_P_IDENT && (warp_pos_x_left - 32) >= (int64_t) x_seq_off && (warp_pos_y_left - 32) >= (int64_t) y_seq_off)
	while(score > 0 && (warp_pos_x_left - 32) >= (int32_t) x_seq_off && (warp_pos_y_left - 32) >= (int32_t) y_seq_off)
	{
		
		warp_pos_x_left -= 32;
		warp_pos_y_left -= 32;
		thre_pos_x = warp_pos_x_left + threadIdx.x;
		thre_pos_y = warp_pos_y_left + threadIdx.x;

		
		char v_x = seq_x[thre_pos_x - (int32_t) x_seq_off];
		char v_y = seq_y[thre_pos_y - (int32_t) y_seq_off];

		
		//if(blockIdx.x == 0) printf("T%d -> %c - %c\n", threadIdx.x, v_x, v_y);
		

		//cell_score = (v_x == v_y && v_x != '\0' && v_y != '\0');
		cell_score = (v_x == v_y);

		for (int offset = 16; offset > 0; offset = offset >> 1)
			cell_score += __shfl_down_sync(0xFFFFFFFF, cell_score, offset);

		
		int idents = __shfl_sync(0xFFFFFFFF, cell_score, 0);

        //if(threadIdx.x == 0) printf("BLOCK %d %d score this round: %d\n", blockIdx.x, counter++, idents);
		score = score + (int32_t) idents;
        total_idents += (int32_t) idents;
		score = score - (int32_t) (32 - idents);
		p_ident = ((100 * total_idents) / (int) (hp1 - warp_pos_x_left + 16));
		if(score > best_score && p_ident >= MIN_P_IDENT){ best_score = score; best_offset_left = warp_pos_x_left; }
	
        //if(threadIdx.x == 0) printf("BLOCK %d %d yea current pident to left %d positions: [start: %u and moved to %u]\n", blockIdx.x, counter++, p_ident, hp1, warp_pos_x_left);	
		
		

	}
	
	
	
	//if(threadIdx.x==0) printf("BLOCK %d %d end left al\n", blockIdx.x, counter++);

	// RIGHT ALIGNMENT
	int32_t warp_pos_x_right = hp1 + 32;
	int32_t warp_pos_y_right = (int32_t) h_p2[blockIdx.x] + 32;
	uint32_t best_offset_right = (uint32_t) warp_pos_x_right;
	score = 32;
	best_score = 32;
	p_ident = 100;
    total_idents = 16;

	
	//while(p_ident > MIN_P_IDENT && (warp_pos_x_right + 32) < (int64_t) x_lim && (warp_pos_y_right + 32) < (int64_t) y_lim)
	while(score > 0 && (warp_pos_x_right + 32) < (int32_t) x_lim && (warp_pos_y_right + 32) < (int32_t) y_lim)
	{
		thre_pos_x = warp_pos_x_right + threadIdx.x;
		thre_pos_y = warp_pos_y_right + threadIdx.x;
		char v_x = seq_x[thre_pos_x - (int32_t) x_seq_off];
		char v_y = seq_y[thre_pos_y - (int32_t) y_seq_off];

		//cell_score = (v_x == v_y && v_x != '\0' && v_y != '\0');
		cell_score = (v_x == v_y);

		for (int offset = 16; offset > 0; offset = offset >> 1)
			cell_score += __shfl_down_sync(0xFFFFFFFF, cell_score, offset);
		
		int idents = __shfl_sync(0xFFFFFFFF, cell_score, 0);
		score = score + (int32_t) idents;
        total_idents += (int32_t) idents;
		score = score - (int32_t) (32 - idents);
		p_ident = (int) ((100 * total_idents) / (int32_t) (warp_pos_x_right - (hp1) + 16));

		warp_pos_x_right += 32;
		warp_pos_y_right += 32; 

        //if(threadIdx.x == 0) printf("BLOCK %d %d yea current pident to right %d positions: [start: %u and moved to %u]\n", blockIdx.x, counter++, p_ident, h_p2[blockIdx.x] + 32, warp_pos_x_right);
		if(score > best_score && p_ident >= MIN_P_IDENT){ best_score = score; best_offset_right = warp_pos_x_right; }

	}
	
	//if(threadIdx.x==0) printf("BLOCK %d %d end right al\n", blockIdx.x, counter++);
	

	// Save at the end
	if(threadIdx.x == 0){
		left_offset[blockIdx.x] = hp1 - (uint32_t) best_offset_left;
        //printf("BLOCK %d %d left offset:  %u\n", blockIdx.x, counter++, left_offset[blockIdx.x]);
		//right_offset[blockIdx.x] = (uint64_t) (best_offset_right + 32) - h_p1[blockIdx.x];
		right_offset[blockIdx.x] = (uint32_t) (best_offset_right) - hp1;
        //printf("BLOCK %d %d right offset: %u\n", blockIdx.x, counter++, right_offset[blockIdx.x]);
	}

}


__global__ void kernel_register_fast_hash_rotational(uint64_t * hashes, uint64_t * positions, const char * sequence, uint64_t offset) {
	


	int i;
	ULLI hash = 0;

	// Each will do 4 kmers
	// So 32 threads do 32*4 = 128 kmers
	// So last kmer processed starts at position 127+32 = we need 160 bytes
	// 160 / 8 = 20
	

	ULLI value = ((ULLI *)sequence)[threadIdx.x + blockIdx.x * 16 ]; // 16*8 = 128 are the bytes 
	ULLI temp_value = value;
	ULLI segment[2] = {0, 0};
	char byte;
	ULLI bad;
	

	i = 0;

	int index = threadIdx.x >> 1; // This is its ID / 2
	int mod2 = (threadIdx.x & 1); // This is threadIdx.x % 2
	int mod2_times_32 = (mod2 << 5); // This is (threadIdx.x % 2) * 32, i.e. a 0 if threadIdx is even and a 32 if its odd

	hash = 0;
	bad = 0xFFFFFFFFFFFFFFFF;

	// Compute the last kmer so that we can make a collaborative algorithm
	// The last INIT kmer starts on 124 (that is 128-4), located on thread 15 on the fourth byte (124/8 = 15.5, 0.5 * 8 = 4)
	// So we distribute the next bytes to the other threads by shuffle
	// Thread 0 will get bytes from thread (124+0)/8 = 15.5 -> thread 15, 
	// and so will the following 4 threads until thread 16 is reached and so on
	temp_value = __shfl_sync(0xFFFFFFFF, value, (124 + threadIdx.x) >> 3);
	// We fetched the full 8 bytes from the value variable, now we need to select only bytes 124
	// Do so by fetching byte i.e. thread 0 has fetched bytes from ULLI from thread 15
	// That is 15*8 = 120 to 127 bytes, but it needs to stick to 124
	//int remainder = ((124 + threadIdx.x) & 2) << 3; // This is % 3 and multiply by 8
	// Which results in 4 for thread 0

	byte = (char) (temp_value >> ((threadIdx.x & 7) << 3));
	//byte = (char) (temp_value >> (8*(remainder-1)));
	

	//if(blockIdx.x == 0) printf("%d -> %c (my val: %d)\n", threadIdx.x, byte, (124 + threadIdx.x) >> 3);

	if(byte == 'C') hash += pow4[threadIdx.x];
	if(byte == 'G') hash += pow4_G[threadIdx.x];
	if(byte == 'T') hash += pow4_T[threadIdx.x];

	// Sum segments and store them in second array 
	//sum (ID+1)*4 mod 32 … + 0,1,2,3

	int offset_segment = (((threadIdx.x + 1) << 2) & 31);

	segment[1] += __shfl_sync(0xFFFFFFFF, hash, offset_segment);
	segment[1] = segment[1] << 2;
	segment[1] += __shfl_sync(0xFFFFFFFF, hash, offset_segment + 1);
	segment[1] = segment[1] << 2;
	segment[1] += __shfl_sync(0xFFFFFFFF, hash, offset_segment + 2);
	segment[1] = segment[1] << 2;
	segment[1] += __shfl_sync(0xFFFFFFFF, hash, offset_segment + 3);
	segment[1] = segment[1] << 2;

	hash = 0;
	
	// The 3 is because it is 0011 in binary so it only selects the destination and source thread with the mask
	temp_value = __shfl_sync(0xFFFFFFFF, value, index);

	
	//byte = ((char*) temp_value)[0 + mod2_times_four]; hash = hash << 2;
	byte = (char) (temp_value >> (24 + mod2_times_32)); hash = hash << 2;
	if((char) byte == 'C') hash = hash + 1; if((char) byte == 'G') hash = hash + 2;
	if((char) byte == 'T') hash = hash + 3; if((char) byte == 'N') bad = 0;

	byte = (char) (temp_value >> (16 + mod2_times_32)); hash = hash << 2;
	if((char) byte == 'C') hash = hash + 1; if((char) byte == 'G') hash = hash + 2;
	if((char) byte == 'T') hash = hash + 3; if((char) byte == 'N') bad = 0;

	byte = (char) (temp_value >> (8 + mod2_times_32)); hash = hash << 2;
	if((char) byte == 'C') hash = hash + 1; if((char) byte == 'G') hash = hash + 2;
	if((char) byte == 'T') hash = hash + 3; if((char) byte == 'N') bad = 0;

	byte = (char) (temp_value >> (0 + mod2_times_32)); hash = hash << 2;
	if((char) byte == 'C') hash = hash + 1; if((char) byte == 'G') hash = hash + 2;
	if((char) byte == 'T') hash = hash + 3; if((char) byte == 'N') bad = 0;

	segment[0] = hash;

	// To this point we have calculated a 4-byte segment in each thread (along with the complete hash of initial-last kmer (no. 124))
	// Now we can begin the "rotations"
	// Each thread will receive the hash value of the next thread

	
	//offset_segment = 0 + ((threadIdx.x + 1) >> 5);
	
	// THis results in a local (global) load because offset is not known at compile time
	// I think the rotational algorithm is better suited for shared memory?
	// With the IFs it doesnt, because it knows the memory address (thread number does not matter)
	
	if(threadIdx.x < 31){

		hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[0], (threadIdx.x + 1) & 31)) << (8)); // 64 
		hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[0], (threadIdx.x + 2) & 31)) << (16));
		hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[0], (threadIdx.x + 3) & 31)) << (24));
		hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[0], (threadIdx.x + 4) & 31)) << (32));
		hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[0], (threadIdx.x + 5) & 31)) << (40));
		hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[0], (threadIdx.x + 6) & 31)) << (48));
		hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[0], (threadIdx.x + 7) & 31)) << (56));

	}else{
		hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[1], (threadIdx.x + 1) & 31)) << (8)); // 64 
		hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[1], (threadIdx.x + 2) & 31)) << (16));
		hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[1], (threadIdx.x + 3) & 31)) << (24));
		hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[1], (threadIdx.x + 4) & 31)) << (32));
		hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[1], (threadIdx.x + 5) & 31)) << (40));
		hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[1], (threadIdx.x + 6) & 31)) << (48));
		hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[1], (threadIdx.x + 7) & 31)) << (56));
	}
	
		
	/*
	hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[offset_segment], (threadIdx.x + 1) & 31)) << (8)); // 64 
	hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[offset_segment], (threadIdx.x + 2) & 31)) << (16));
	hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[offset_segment], (threadIdx.x + 3) & 31)) << (24));
	hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[offset_segment], (threadIdx.x + 4) & 31)) << (32));
	hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[offset_segment], (threadIdx.x + 5) & 31)) << (40));
	hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[offset_segment], (threadIdx.x + 6) & 31)) << (48));
	hash = hash + ((__shfl_sync(0xFFFFFFFF, segment[offset_segment], (threadIdx.x + 7) & 31)) << (56));
	*/
	
	
	//printf("PREV at %"PRIu64" -> %.32s\n", 0, &sequence[0]);
	
	//table[threadIdx.x + 32*i + 192 * blockIdx.x] = hash & bad;
	hashes[threadIdx.x + 128 * blockIdx.x] = hash & bad;
	positions[threadIdx.x + 128 * blockIdx.x] = (threadIdx.x + blockIdx.x * 128 + offset) | (~bad);
	//table[threadIdx.x + 96 * blockIdx.x] = hash & bad;
	

	unsigned kmer_start = (32 + (threadIdx.x << 2)); // 4 because of 4 kmers per thread
	unsigned int_pos = kmer_start >> 3; // 8 because bytes per register

	for(i=1; i<4; i++){
		
		bad = 0xFFFFFFFFFFFFFFFF;
		temp_value = __shfl_sync(0xFFFFFFFF, value, int_pos);
		byte = (char) (temp_value >> ((kmer_start & 7) << 3));
		++kmer_start;
		int_pos = kmer_start >> 3;

		// Remember - the kmer is not well built right now but first focus on performance

		
		hash = hash << 2;

		
		if((char) byte == 'C') hash = hash + 1;
		if((char) byte == 'G') hash = hash + 2;
		if((char) byte == 'T') hash = hash + 3;
		if((char) byte == 'N') bad = 0;
		

		/*
		hash += (byte & 6) >> 1;
		if(byte == 'N') bad = 0;
		*/

		//table[threadIdx.x + blockIdx.x * blockDim.x*8 + blockDim.x * k] = hash & bad;
		hashes[threadIdx.x + (i << 5) + 128 * blockIdx.x] = hash & bad;
		positions[threadIdx.x + (i << 5) + 128 * blockIdx.x] = (threadIdx.x + (i << 5) + blockIdx.x * 128 + offset) | (~bad);

		
	}

	//if(threadIdx.x == 0 && blockIdx.x == 0) printf("POST at %"PRIu64" -> %.32s @ %"PRIu64"\n", threadIdx.x + (32) + 128 * blockIdx.x, &sequence[threadIdx.x + (32) + 128 * blockIdx.x], hashes[threadIdx.x + (32) + 128 * blockIdx.x]);
	
}

__global__ void kernel_index_global32(uint64_t * hashes, uint32_t * positions, const char * sequence, uint32_t offset) {
		

	uint64_t k, hash = 0;
		
	uint64_t bad = 0xFFFFFFFFFFFFFFFF;

	for(k=0; k<32; k++){

		char c = sequence[threadIdx.x + k + blockIdx.x * blockDim.x];
		//if(threadIdx.x == 0) printf("%c", c);
		//char c = sequence[0];

		if(c == 'A') hash += 0;
		if(c == 'C') hash += pow4[k];
		if(c == 'G') hash += pow4_G[k];
		if(c == 'T') hash += pow4_T[k];
		if(c == 'N') bad = 0;
		
	}

	// [0 - 32] * [0-N]* [32]
	hashes[threadIdx.x + blockIdx.x * blockDim.x] = hash & bad;
	//hashes[0] = hash & bad;
	positions[threadIdx.x + blockIdx.x * blockDim.x] = (threadIdx.x + blockIdx.x * blockDim.x + offset) | (~bad);
}

__global__ void kernel_index_global_shared(uint64_t * hashes, uint32_t * positions, const char * sequence, uint32_t offset) {
		

	uint64_t k, hash = 0;
		
	uint64_t bad = 0xFFFFFFFFFFFFFFFF;

	for(k=0; k<blockDim.x; k++){

		char c = sequence[threadIdx.x + k + blockIdx.x * blockDim.x];
		//if(threadIdx.x == 0) printf("%c", c);
		//char c = sequence[0];

		if(c == 'A') hash += 0;
		if(c == 'C') hash += pow4[k];
		if(c == 'G') hash += pow4_G[k];
		if(c == 'T') hash += pow4_T[k];
		if(c == 'N') bad = 0;
		
	}

	// [0 - 32] * [0-N]* [32]
	hashes[threadIdx.x + blockIdx.x * blockDim.x] = hash & bad;
	//hashes[0] = hash & bad;
	positions[threadIdx.x + blockIdx.x * blockDim.x] = (threadIdx.x + blockIdx.x * blockDim.x + offset) | (~bad);
}

__global__ void kernel_reverse_complement(const char * sequence, char * reverse_sequence, uint32_t seq_len) {
	
	//uint64_t id = (31 - threadIdx.x) + blockIdx.x * blockDim.x;
	uint64_t id = threadIdx.x + blockIdx.x * blockDim.x;
	

	if(id < seq_len){

		//uint64_t lookup = seq_len - id;
		uint32_t lookup = (seq_len - 1) - id;
		char original = sequence[id];
		char complement = 'N';
		if(original == 'A') complement = 'T';
		if(original == 'C') complement = 'G';
		if(original == 'G') complement = 'C';
		if(original == 'T') complement = 'A';

		reverse_sequence[lookup] = complement;
	}
}

